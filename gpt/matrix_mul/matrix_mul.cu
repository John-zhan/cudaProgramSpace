#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 16384  // 矩阵尺寸 N x N

__global__ void matrixMulKernel(float *A, float *B, float *C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;  // 行号
    int col = blockIdx.x * blockDim.x + threadIdx.x;  // 列号

    if (row < n && col < n) {
        float sum = 0.0f;
        for (int k = 0; k < n; ++k)
            sum += A[row * n + k] * B[k * n + col];
        C[row * n + col] = sum;
    }
}

int main() {
    size_t size = N * N * sizeof(float);

    // 主机内存分配
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // 初始化输入矩阵
    for (int i = 0; i < N * N; ++i) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    // 设备内存分配
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // 拷贝数据到 GPU
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // 启动 kernel
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + 15) / 16, (N + 15) / 16);
    matrixMulKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // 等待完成
    hipDeviceSynchronize();

    // 拷贝结果回主机
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // 验证结果
    printf("C[0][0] = %f\n", h_C[0]);

    // 清理
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
